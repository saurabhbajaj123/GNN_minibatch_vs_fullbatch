#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
// includes

#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>
#include <memory>
#include <iostream>
#include <vector>
#include <unistd.h>

#define cudaCheckError()                                       \
  {                                                            \
    hipError_t e = hipGetLastError();                        \
    if (e != hipSuccess) {                                    \
      printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, \
             hipGetErrorString(e));                           \
      exit(EXIT_FAILURE);                                      \
    }                                                          \
  }


#define WARP_SIZE 32
#define WARP_GROUP  4
__device__ int find(const int64_t *offsets, const int device_count,
    const int64_t index)
{
int i = 1;
for (i = 1; i < device_count; i++) {
if (index < offsets[i]) { return i - 1; }
}
return device_count - 1;
}
__global__ void quiver_tensor_gather(float **dev_ptrs, const int64_t *offsets,
                     const int device_count,
                     const int64_t *indices, int indice_length,
                     float *res, const int stride, const int* access_book)
{

    //
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int step = gridDim.x * blockDim.x;

    // each warp take charge of one-feature copy
    unsigned int warp_id = tid / WARP_SIZE;
    unsigned int warp_step = step / WARP_SIZE;

    unsigned int warp_start = warp_id;
    unsigned int thread_start = tid % WARP_SIZE;

    int64_t dev_index = 0;
    int64_t dev_offset = 0;
    float *dev_ptr;
    int64_t src_copy_start = 0;
    int64_t dst_copy_start = 0;

    unsigned int local_start = thread_start;
    while (warp_start < indice_length) {
        local_start = thread_start;
        dev_index = find(offsets, device_count, indices[warp_start]);
        // we only copy data from reachable device
        if(access_book[dev_index] == 1){
        dev_ptr = dev_ptrs[dev_index];
        dev_offset = indices[warp_start] - offsets[dev_index];
        src_copy_start = dev_offset * stride;
        dst_copy_start = warp_start * stride;
        for (; local_start < stride; local_start += WARP_SIZE) {
            res[dst_copy_start + local_start] =
                dev_ptr[src_copy_start + local_start];
        }
        }
        warp_start += warp_step;
    }
}

int main(){
    int numGPUs, numElems =  40000;
    hipGetDeviceCount(&numGPUs);
    int current_device = 0;
    std::cout<<"device count = " << numGPUs <<std::endl;
    std::vector<float *> buffers(numGPUs);
    std::vector<int64_t> offset_host;
    std::vector<int> access_book;
    std::vector<hipStream_t> stream(numGPUs);


    std::cout<<"offset_host initialization finished " <<offset_host.size() <<std::endl;

    std::vector<int64_t> indices_host;
    int offset_val = 0;
    for(int index = 0; index < numElems; index++){
        indices_host.push_back(rand() % (numElems * numGPUs));
        offset_host.push_back(offset_val);
        offset_val += numElems;
    }

    std::cout<<"indices_host initialization finished " <<indices_host.size() <<std::endl;

    // P2P Initilization
    for (int i = 0; i < numGPUs; i++) {
        hipSetDevice(i);
        for (int j = i + 1; j < numGPUs; j++) {
          int access = 0;
          hipDeviceCanAccessPeer(&access, i, j);
          if (access) {
            hipSetDevice(i);
            hipDeviceEnablePeerAccess(j, 0);
            cudaCheckError();
            hipSetDevice(j);
            hipDeviceEnablePeerAccess(i, 0);
            cudaCheckError();
          }
        }
    }


    
    
    int64_t* offset_device;
    hipMalloc((void**) &offset_device, sizeof(int64_t) * offset_host.size());
    hipMemcpy(offset_device, &offset_host[0], sizeof(int64_t) * offset_host.size(), hipMemcpyHostToDevice);
    cudaCheckError();

    int64_t* indices_device;
    hipMalloc((void**) &indices_device, sizeof(int64_t) * indices_host.size());
    hipMemcpy(indices_device, &indices_host[0], sizeof(int64_t) * indices_host.size(), hipMemcpyHostToDevice);
    cudaCheckError();

    float* res_device;
    float* res_host = (float*) malloc(sizeof(float) * numElems);
    hipMalloc((void**) &res_device, sizeof(float) * numElems);
    cudaCheckError();


    for (int d = 0; d < numGPUs; d++) {
         hipSetDevice(d);
         hipMalloc((void**) &buffers[d], numElems * sizeof(float));
         hipMemset(buffers[d], 0, numElems * sizeof(float));
         cudaCheckError();
    }


    hipSetDevice(current_device);
    for (int i = 0; i < numGPUs; i++) {
        
        hipDeviceCanAccessPeer(&access, current_device, i);
        if(access || i ==  current_device){
            access_book.push_back(1);
        }else{
            access_book.push_back(0);
        }
    }


    hipPointerAttribute_t attributes;
    hipPointerGetAttributes(&attributes, (void*) buffers[1]);
    std::cout<< "check device " << attributes.device << " check device pointer" << attributes.devicePointer<<std::endl;
    
    hipPointerGetAttributes(&attributes,  attributes.devicePointer);
    std::cout<< "check device " << attributes.device << " check device pointer" << attributes.devicePointer<<std::endl;


    float ** buffers_device;
    hipMalloc((void ***) &buffers_device, sizeof(float*) * numGPUs);
    hipMemcpy(buffers_device, &buffers[0], sizeof(float*) * buffers.size(), hipMemcpyHostToDevice);
    cudaCheckError();
    
    int* access_book_device;
    hipMalloc((void **) &access_book_device, sizeof(int) * access_book.size());
    hipMemcpy(buffers_device, &buffers[0], sizeof(float*) * buffers.size(), hipMemcpyHostToDevice);
    cudaCheckError();
    

    std::cout<<"all data initialization finished " <<std::endl;


    quiver_tensor_gather<<<1024, 512>>>(buffers_device, offset_device, numGPUs, indices_device, numElems, res_device, 1);
    hipDeviceSynchronize();
    hipDeviceSynchronize();
    cudaCheckError();

    std::cout<<"test finished " <<std::endl;
}
